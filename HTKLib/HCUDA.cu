#include "hip/hip_runtime.h"
/* ----------------------------------------------------------- */
/*                                                             */
/*                          ___                                */
/*                       |_| | |_/   SPEECH                    */
/*                       | | | | \   RECOGNITION               */
/*                       =========   SOFTWARE                  */
/*                                                             */
/*                                                             */
/* ----------------------------------------------------------- */
/* developed at:                                               */
/*                                                             */
/*           Machine Intelligence Laboratory                   */
/*           Department of Engineering                         */
/*           University of Cambridge                           */
/*           http://mi.eng.cam.ac.uk/                          */
/*                                                             */
/* author:                                                     */
/*           Chao Zhang <cz277@cam.ac.uk>                      */
/*                                                             */
/* ----------------------------------------------------------- */
/*           Copyright: Cambridge University                   */
/*                      Engineering Department                 */
/*            2013-2015 Cambridge, Cambridgeshire UK           */
/*                      http://www.eng.cam.ac.uk               */
/*                                                             */
/*   Use of this software is governed by a License Agreement   */
/*    ** See the file License for the Conditions of Use  **    */
/*    **     This banner notice must not be removed      **    */
/*                                                             */
/* ----------------------------------------------------------- */
/*                File: HCUDA.h   CUDA utilities               */
/* ----------------------------------------------------------- */


#ifdef __cplusplus
extern "C" {
#endif

const char *hcuda_version = "!HVER!HCUDA:   3.5.0 [CUED 12/10/15]";
const char *hcuda_vc_id = "$Id: HCUDA.cu,v 1.0 2015/10/12 12:07:23 cz277 Exp $";

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "HCUDA.h"
#include "HShell.h"
#include "HMem.h"
#include "HMath.h"
#include "config.h"


/* --------------------------- Trace Flags ------------------------ */

#define CEIL(x,y) (((x)+(y)-1) / (y))

/* --------------------------- Trace Flags ------------------------ */
#define T_TOP 0001                              /* Top Level tracing */

static ConfParam *cParm[MAXGLOBS];              /* config parameters */
static int nParm = 0;

static int GPUDevId = -1;                       /*  */
static Boolean GPUInit = FALSE;                 /*  */
static const char *GPUIdEnvVar = "";                  /*  */
hipblasHandle_t handle;				/*  */
static size_t GPUMemUsed = 0;			/*  */

/* ----------------------- Device Management ---------------------- */

/*  */
static void ShowAllGPUs(void) {
    int nGPU, i;
    hipError_t error;
    hipDeviceProp_t prop;
    /*CUResult result;*/

    error = hipGetDeviceCount(&nGPU);    
    if (error != hipSuccess) 
        HError(8800, (char *)"ShowAllGPUs: %s", hipGetErrorString(error)); 
    if (nGPU == 0) 
        HError(8820, (char *)"ShowAllGPUs: No GPU device");
    for (i = 0; i < nGPU; ++i) {
        error = hipGetDeviceProperties(&prop, i);
        if (error != hipSuccess) 
            HError(8800, (char *)"ShowAllGPUs: %s", hipGetErrorString(error));
        printf("GPU %d: %s, %luMB, SM = %d.%d", i, prop.name, prop.totalGlobalMem / 1048576, prop.major, prop.minor);
        if (GPUDevId == i)
            printf(" [Selected]");
        printf("\n");
    }
}

/* To check CUDA requirement */
static void CheckCUDAReq(hipDeviceProp_t *prop)
{
    int driverVer;
    int runtimeVer;
    int cublasVer;
    hipError_t error;    
    hipblasStatus_t status;
    
    error = hipDriverGetVersion(&driverVer);
    if (error != hipSuccess) 
        HError(8800, (char *)"CheckCUDAReq: %s", hipGetErrorString(error));
    if (driverVer < MINCUDAVER) 
        HError(8800, (char *)"CheckCUDAReq: CUDA driver version %d is lower than the minimum required version %d", driverVer, MINCUDAVER);

    error = hipRuntimeGetVersion(&runtimeVer);
    if (error != hipSuccess) 
        HError(8800, (char *)"CheckCUDAReq: %s", hipGetErrorString(error));
    if (runtimeVer < MINCUDAVER) 
        HError(8800, (char *)"CheckCUDAReq: CUDA runtime version %d is lower than the minimum required version %d", runtimeVer, MINCUDAVER);

    status = cublasGetVersion(handle, &cublasVer);
    if (status != HIPBLAS_STATUS_SUCCESS) 
        HError(8800, (char *)"CheckCUDAReq: Fail to get CUBLAS library version");
    if (cublasVer < MINCUDAVER) 
        HError(8800, (char *)"CheckCUDAReq: CUBLAS library version %d is lower than the minimum required version %d", cublasVer, MINCUDAVER);

    if (prop->major <= MINMAJORSMARCH && prop->minor <= MINMINORSMARCH) 
        HError(8800, (char *)"CheckCUDAReq: SM architecture is lower than the minimum requirement, %d.%d", MINMAJORSMARCH, MINMINORSMARCH);

    printf("CUDA driver version %d\n", driverVer);
    printf("CUDA runtime version %d\n", runtimeVer);
    printf("CUBLAS library version %d\n", cublasVer);
}

/* Initialize the GPU device. It first loads the GPU device
   from the config file. Then
*/
void InitCUDA(void)
{
    ConfParam *cpVal;

    Register((char *)hcuda_version, (char *)hcuda_vc_id);

    /* load parameters from the config file */
    nParm = GetConfig((char *)"HCUDA", TRUE, cParm, MAXGLOBS);
    if (nParm > 0) {
        if (GetConfAny(cParm, nParm, (char *)"GPUID", &cpVal)) {
            if (cpVal->kind == IntCKind) 
                GPUDevId = cpVal->val.i;
            else if (cpVal->kind == StrCKind) 
                GPUIdEnvVar = CopyString(&gcheap, cpVal->val.s);
            else 
                HError(8820, (char *)"InitCUDA: Unknown GPUID value kind");
            /*strcpy(buf, cpVal->val.s);
            GPUIdEnvVar = (char *) New(&gcheap, sizeof(char) * strlen(buf));
            strcpy(GPUIdEnvVar, buf);*/
        }
    }
}

/*  */
void StartCUDA(void) {
    char *envVar;
    hipError_t error;
    hipblasStatus_t status;
    hipDeviceProp_t prop;

    /* initialize the library and device */
    if (!GPUInit) {
        /* select a device */
        if (strcmp(GPUIdEnvVar, "") != 0) { /* use env variable */
            envVar = getenv(GPUIdEnvVar);
            if (envVar == NULL) {
                HError(-8821, (char *)"InitCUDA: Environment variable %s not defined, reset to use GPU 0\n", GPUIdEnvVar);
                GPUDevId = 0;
            }
            else {
                GPUDevId = atoi(envVar);
            }
        }
        if (GPUDevId < 0) {
            error = hipChooseDevice(&GPUDevId, &prop);
            if (error != hipSuccess) 
                HError(8800, (char *)"InitCUDA: %s", hipGetErrorString(error));
        }
        error = hipSetDevice(GPUDevId);
        if (error != hipSuccess) 
            HError(8800, (char *)"InitCUDA: %s", hipGetErrorString(error));
        error = hipGetDeviceProperties(&prop, GPUDevId);
        if (error != hipSuccess) 
            HError(8800, (char *)"InitCUDA: %s", hipGetErrorString(error));
        /* initiate CUBLAS */
        status = hipblasCreate(&handle);
        if (status != HIPBLAS_STATUS_SUCCESS) 
            HError(8800, (char *)"InitCUDA: Fail to initialise CUBLAS");
        /* check version */
        CheckCUDAReq(&prop);
        /* set GPUInit flag */
        GPUInit = TRUE;
        /* show devices */
        ShowAllGPUs();
    }
    else {
        printf("InitCUDA: GPU device %d already initialised", GPUDevId);
    }
    printf("\n");
}

/*  */
void StopCUDA(void) {
    if (GPUInit) {
        /* destroy the context on the GPU */
        hipblasDestroy(handle);
        /* shutdown CUBLAS */
        hipDeviceReset();
        /* reset GPU IDs and the flag */
        GPUDevId = -1;
        GPUInit = FALSE;
    }
    else {
        printf("StopCUDA: GPU device has already stopped");
    }
}

/* --------------------------- Trace Flags ------------------------ */

__global__ void HKern_SetNSegment(NFloat val, NFloat *segPtr, int segLen) {
    int pos;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < segLen) {
        segPtr[pos] = val;
    }
}

__global__ void HKern_ScaledSelfAddNSegment(NFloat *rhPtr, int segLen, NFloat scale, NFloat *lhPtr) {
    int pos;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < segLen) {
        lhPtr[pos] = scale * lhPtr[pos] + rhPtr[pos];
    }
}

__global__ void HKern_DupNSegment(NFloat *srcPtr, int segLen, NFloat *dstPtr, int times) {
    int srcPos, dstPos;
    
    dstPos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (dstPos < segLen * times) {
        srcPos = dstPos % segLen;
        dstPtr[dstPos] = srcPtr[srcPos];
    }
}

__global__ void HKern_SubNSegment(NFloat *lhPtr, NFloat *rhPtr, int segLen, NFloat *resPtr) {
    int pos;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < segLen) {
        resPtr[pos] = lhPtr[pos] - rhPtr[pos];
    }
}

__global__ void HKern_MulNSegment(NFloat *lhPtr, NFloat *rhPtr, int segLen, NFloat *resPtr) {
    int pos;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < segLen) {
        resPtr[pos] = lhPtr[pos] * rhPtr[pos];
    }
}

/* cz277 - pact */
__global__ void HKern_ApplyAffineAct(NFloat *srcPtr, int row, int col, NFloat *scalePtr, NFloat *shiftPtr, NFloat *dstPtr) {
    int pos, rowIdx, colIdx;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    rowIdx = pos / col;
    colIdx = pos % col;
    if (rowIdx < row) {
        dstPtr[pos] = scalePtr[colIdx] * srcPtr[pos] + shiftPtr[colIdx];
    }
}

/* cz277 - pact */
__global__ void HKern_ApplyDAffineAct(NFloat *srcPtr, int row, int col, NFloat *scalePtr, NFloat *shiftPtr, NFloat *dstPtr) {
    int pos, rowIdx, colIdx;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    rowIdx = pos / col;
    colIdx = pos % col;
    if (rowIdx < row) {
        dstPtr[pos] = scalePtr[colIdx];
    }
}


/* cz277 - pact */
__global__ void HKern_ApplyTrAffineAct(NFloat *errPtr, NFloat *actPtr, int row, int col, NFloat *scalePtr, NFloat *shiftPtr, Boolean accFlag, NFloat *dScalePtr, NFloat *dShiftPtr) {
    extern __shared__ NFloat tmpPtr[];
    int thdIdx, colIdx, thdNum, base, idx, incr, pos, step, off = THREADPERBLOCK;

    thdIdx = threadIdx.x;       /* num threads per block */
    colIdx = blockIdx.x;        /* block index, i.e., row num (minibatch size)  */
    thdNum = min(blockDim.x, row);
    if (thdIdx < thdNum && colIdx < col) {
        /* collect the sums for the groups (and transpose the matrix) */
        tmpPtr[thdIdx] = 0.0;		/* dScale */
        tmpPtr[off + thdIdx] = 0.0;	/* dShift */
        base = colIdx;
        idx = thdIdx;
        pos = base + idx * col;
        step = thdNum * col;
        while (idx < row) {
            tmpPtr[thdIdx] += errPtr[pos] * actPtr[pos];
            tmpPtr[off + thdIdx] += errPtr[pos];
            pos += step;
            idx += thdNum;
        }
        __syncthreads();
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx) {
                    tmpPtr[thdIdx] += tmpPtr[pos];
                    tmpPtr[off + thdIdx] += tmpPtr[off + pos];
                }
            }
            __syncthreads();
        }

        /* copy the value to dstPtr */
        if (thdIdx == 0) {
            if (accFlag == FALSE) {
                dScalePtr[colIdx] = 0.0;
                dShiftPtr[colIdx] = 0.0;
            }
            dScalePtr[colIdx] += tmpPtr[0];
            dShiftPtr[colIdx] += tmpPtr[off + 0];
        }
    }
}

/* cz277 - laf */
__global__ void HKern_AccMeanNSegment(NFloat *valPtr, int row, int col, NFloat tSamp, NFloat *meanPtr) {
        extern __shared__ NFloat tmpPtr[];
    int thdIdx, colIdx, thdNum, base, idx, incr, pos, step;

    thdIdx = threadIdx.x;       /* num threads per block */
    colIdx = blockIdx.x;        /* block index, i.e., row num (minibatch size)  */
    thdNum = min(blockDim.x, row);
    if (thdIdx < thdNum && colIdx < col) {
        /*base = colIdx;*/
        /* collect the sums for the groups (and transpose the matrix) */
        tmpPtr[thdIdx] = 0.0;/*srcPtr[base + idx * col];*/
        base = colIdx;
        idx = thdIdx;
        pos = base + idx * col;
        step = thdNum * col;
        while (idx < row) {
            tmpPtr[thdIdx] += valPtr[pos] / tSamp;
            pos += step;
            idx += thdNum;
        }
        __syncthreads();
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx) {
                    tmpPtr[thdIdx] += tmpPtr[pos];
                }
            }
            __syncthreads();
        }

        /* copy the value to dstPtr */
        if (thdIdx == 0) {
            meanPtr[colIdx] += tmpPtr[0];
        }
    }
}

/* cz277 - laf */
__global__ void HKern_AccVarianceNSegment(NFloat *valPtr, int row, int col, NFloat tSamp, NFloat *meanPtr, NFloat *varPtr) {
    extern __shared__ NFloat tmpPtr[];
    int thdIdx, colIdx, thdNum, base, idx, incr, pos, step;

    thdIdx = threadIdx.x;       /* num threads per block */
    colIdx = blockIdx.x;        /* block index, i.e., row num (minibatch size)  */
    thdNum = min(blockDim.x, row);
    if (thdIdx < thdNum && colIdx < col) {
        /*base = colIdx;*/
        /* collect the sums for the groups (and transpose the matrix) */
        tmpPtr[thdIdx] = 0.0;/*srcPtr[base + idx * col];*/
        base = colIdx;
        idx = thdIdx;
        pos = base + idx * col;
        step = thdNum * col;
        while (idx < row) {
            tmpPtr[thdIdx] += pow(valPtr[pos] - meanPtr[colIdx], 2) / tSamp;
            pos += step;
            idx += thdNum;
        }
        __syncthreads();
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx) {
                    tmpPtr[thdIdx] += tmpPtr[pos];
                }
            }
            __syncthreads();
        }

        /* copy the value to dstPtr */
        if (thdIdx == 0) {
            varPtr[colIdx] += tmpPtr[0];
        }
    }
}


/* cz277 - pact */
__global__ void HKern_ApplyParmReLUAct(NFloat *srcPtr, int row, int col, NFloat *posPtr, NFloat *negPtr, NFloat *dstPtr) {
    int pos, rowIdx, colIdx;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    rowIdx = pos / col;
    colIdx = pos % col;
    if (rowIdx < row) {
        if (srcPtr[pos] > 0.0)
            dstPtr[pos] = posPtr[colIdx] * srcPtr[pos];
        else
            dstPtr[pos] = negPtr[colIdx] * srcPtr[pos];
    }
}

/* cz277 - pact */
__global__ void HKern_ApplyDParmReLUAct(NFloat *inpPtr, int row, int col, NFloat *posPtr, NFloat *negPtr, NFloat *dstPtr) {
    int pos, rowIdx, colIdx;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    rowIdx = pos / col;
    colIdx = pos % col;
    if (rowIdx < row) {
        if (inpPtr[pos] > 0.0)
            dstPtr[pos] = posPtr[colIdx];
        else
            dstPtr[pos] = negPtr[colIdx];
    }
}


/* cz277 - pact */
__global__ void HKern_ApplyTrParmReLUAct(NFloat *errPtr, NFloat *inpPtr, int row, int col, Boolean accFlag, NFloat *dPosPtr, NFloat *dNegPtr) {
    extern __shared__ NFloat tmpPtr[];
    int thdIdx, colIdx, thdNum, base, idx, incr, pos, step, off = THREADPERBLOCK;

    thdIdx = threadIdx.x;       /* num threads per block */
    colIdx = blockIdx.x;        /* block index, i.e., row num (minibatch size)  */
    thdNum = min(blockDim.x, row);
    if (thdIdx < thdNum && colIdx < col) {
        /* collect the sums for the groups (and transpose the matrix) */
        tmpPtr[thdIdx] = 0.0;		/* alpha */
        tmpPtr[off + thdIdx] = 0.0;	/* beta */
        base = colIdx;
        idx = thdIdx;
        pos = base + idx * col;
        step = thdNum * col;
        while (idx < row) {
            if (inpPtr[pos] > 0.0)
                tmpPtr[thdIdx] += errPtr[pos] * inpPtr[pos];
            else
                tmpPtr[off + thdIdx] += errPtr[pos] * inpPtr[pos];
            pos += step;
            idx += thdNum;
        }
        __syncthreads();
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx) {
                    tmpPtr[thdIdx] += tmpPtr[pos];
                    tmpPtr[off + thdIdx] += tmpPtr[off + pos];
                }
            }
            __syncthreads();
        }

        /* copy the value to dstPtr */
        if (thdIdx == 0) {
            if (accFlag == FALSE) {
                dPosPtr[colIdx] = 0.0;
                dNegPtr[colIdx] = 0.0;
            }
            dPosPtr[colIdx] += tmpPtr[0];
            dNegPtr[colIdx] += tmpPtr[off + 0];
        }
    }
}


/* cz277 - laf */
__global__ void HKern_ApplyPReLUAct(NFloat *srcPtr, int row, int col, NFloat *scalePtr, NFloat *dstPtr) {
    int pos, rowIdx, colIdx;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    rowIdx = pos / col;
    colIdx = pos % col;
    if (rowIdx < row) {
        if (srcPtr[pos] > 0.0)
            dstPtr[pos] = scalePtr[colIdx] * srcPtr[pos];
        else
            dstPtr[pos] = 0.0;
    }
}

/* cz277 - pact */
__global__ void HKern_ApplyDPReLUAct(NFloat *srcPtr, int row, int col, NFloat *scalePtr, NFloat *dstPtr) {
    int pos, rowIdx, colIdx;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    rowIdx = pos / col;
    colIdx = pos % col;
    if (rowIdx < row) {
        if (scalePtr[colIdx] != 0.0 && srcPtr[pos] / scalePtr[colIdx] > 0.0)
            dstPtr[pos] = scalePtr[colIdx];
        else
            dstPtr[pos] = 0.0;
    }
}

/* cz277 - pact */
__global__ void HKern_ApplyTrPReLUAct(NFloat *errPtr, NFloat *srcPtr, int row, int col, NFloat *scalePtr, Boolean accFlag, NFloat *dScalePtr) {
    extern __shared__ NFloat tmpPtr[];
    int thdIdx, colIdx, thdNum, base, idx, incr, pos, step;
    NFloat act;

    thdIdx = threadIdx.x;       /* num threads per block */
    colIdx = blockIdx.x;        /* block index, i.e., row num (minibatch size)  */
    thdNum = min(blockDim.x, row);
    if (thdIdx < thdNum && colIdx < col) {
        /* collect the sums for the groups (and transpose the matrix) */
        tmpPtr[thdIdx] = 0.0;	/*srcPtr[base + idx * col];*/
        base = colIdx;
        idx = thdIdx;
        pos = base + idx * col;
        step = thdNum * col;
        while (idx < row) {
            if (scalePtr[colIdx] != 0.0) {
                act = srcPtr[pos] / scalePtr[colIdx];
                if (act > 0.0)
                    tmpPtr[thdIdx] += errPtr[pos] * act;
            }
            pos += step;
            idx += thdNum;
        }
        __syncthreads();
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx) {
                    tmpPtr[thdIdx] += tmpPtr[pos];
                }
            }
            __syncthreads();
        }

        /* copy the value to dstPtr */
        if (thdIdx == 0) {
            if (accFlag == FALSE)
                dScalePtr[colIdx] = 0.0;
            dScalePtr[colIdx] += tmpPtr[0];
        }
    }
}

__global__ void HKern_ApplyReLUAct(NFloat *srcPtr, int len, NFloat scale, NFloat *dstPtr) {
    int pos;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < len) {
        if (srcPtr != dstPtr && srcPtr[pos] > 0) {
            dstPtr[pos] = srcPtr[pos];
        }
        if (srcPtr[pos] < 0) {
            dstPtr[pos] = srcPtr[pos] * scale;
            /* cz277 - standard ReLU */
            /*dstPtr[pos] = 0.0;*/
        }
    }
}

__global__ void HKern_ApplyDReLUAct(NFloat *srcPtr, int len, NFloat scale, NFloat *dstPtr) {
    int pos;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < len) {
        if (srcPtr[pos] > 0.0) {
            dstPtr[pos] = 1.0;
        }
        else {
            dstPtr[pos] = scale;
            /* cz277 - standard ReLU */
            /*dstPtr[pos] = 0.0;*/
        }
    }
}

__global__ void HKern_ApplyDLinearAct(NFloat *srcPtr, int len, NFloat *dstPtr) {
    int pos;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < len) {
        dstPtr[pos] = 1.0;
    }
}

__global__ void HKern_ApplyLHUCSigmoidAct(NFloat *srcPtr, int row, int col, NFloat *rolePtr, NFloat *dstPtr) {
    int pos, colIdx;
    NFloat floatVal, lhucVal;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < row * col) {
        colIdx = pos % col;
        floatVal = -1.0 * rolePtr[colIdx];
        CHKNFLTEXPE(floatVal)
        lhucVal = 2.0 / (1.0 + exp(floatVal));
        floatVal = -1.0 * srcPtr[pos];
        CHKNFLTEXPE(floatVal)
        dstPtr[pos] = lhucVal * 1.0 / (1.0 + exp(floatVal));
    }
}

__global__ void HKern_ApplyDLHUCSigmoidAct(NFloat *srcPtr, int row, int col, NFloat *rolePtr, NFloat *dstPtr) {
    int pos, colIdx;
    NFloat floatVal, lhucVal;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < row * col) {
        colIdx = pos % col;
        floatVal = -1.0 * rolePtr[colIdx];
        CHKNFLTEXPE(floatVal)
        lhucVal = 2.0 / (1.0 + exp(floatVal));
        floatVal = srcPtr[pos] / lhucVal;
        dstPtr[pos] = srcPtr[pos] * (1.0 - floatVal);
    }
}

__global__ void HKern_ApplyTrLHUCSigmoidActCUDA(NFloat *errPtr, NFloat *actPtr, int row, int col, NFloat *rolePtr, Boolean accFlag, NFloat *dRolePtr) {
    extern __shared__ NFloat tmpPtr[];
    int thdIdx, colIdx, thdNum, base, idx, incr, pos, step;
    NFloat floatVal;

    thdIdx = threadIdx.x;       /* num threads per block */
    colIdx = blockIdx.x;        /* block index, i.e., row num (minibatch size)  */
    thdNum = min(blockDim.x, row);
    if (thdIdx < thdNum && colIdx < col) {
        floatVal = -1.0 * rolePtr[colIdx];
        CHKNFLTEXPE(floatVal)
        floatVal = 0.5 * 2.0 / (1.0 + exp(floatVal));
        /*base = colIdx;*/
        /* collect the sums for the groups (and transpose the matrix) */
        tmpPtr[thdIdx] = 0.0;/*actPtr[base + idx * col];*/
        base = colIdx;
        idx = thdIdx;
        pos = base + idx * col;
        step = thdNum * col;
        while (idx < row) {
            tmpPtr[thdIdx] += errPtr[pos] * actPtr[pos] * (1.0 - floatVal);
            pos += step;
            idx += thdNum;
        }
        __syncthreads();
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx) {
                    tmpPtr[thdIdx] += tmpPtr[pos];
                }
            }
            __syncthreads();
        }

        /* copy the value to dstPtr */
        if (thdIdx == 0) {
            if (accFlag == FALSE)
                dRolePtr[colIdx] = 0.0;
            dRolePtr[colIdx] += tmpPtr[0];
        }
    }
}


__global__ void HKern_ApplyParmSigmoidAct(NFloat *srcPtr, int row, int col, NFloat *etaPtr, NFloat *gammaPtr, NFloat* thetaPtr, NFloat *dstPtr) {
    int pos, colIdx;
    NFloat floatVal;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < row * col) {
        colIdx = pos % col;
        floatVal = (-1.0) * gammaPtr[colIdx] * srcPtr[pos] + thetaPtr[colIdx];
        CHKNFLTEXPE(floatVal)
        dstPtr[pos] = etaPtr[colIdx] / (1.0 + exp(floatVal));
    }
}

__global__ void HKern_ApplyDParmSigmoidAct(NFloat *srcPtr, int row, int col, NFloat *etaPtr, NFloat *gammaPtr, NFloat *thetaPtr, NFloat *dstPtr) {
    int pos, colIdx;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < row * col) {
        colIdx = pos % col;
        if (etaPtr[colIdx] != 0.0)
            dstPtr[pos] = gammaPtr[colIdx] * srcPtr[pos] * (1.0 - srcPtr[pos] / etaPtr[colIdx]);
        else
            dstPtr[pos] = 0.0;
    }
}

__global__ void HKern_ApplyTrParmSigmoidActCUDA(NFloat *errPtr, NFloat *inpPtr, int row, int col, NFloat *etaPtr, NFloat *gammaPtr, NFloat *thetaPtr, Boolean accFlag, NFloat *dEtaPtr, NFloat *dGammaPtr, NFloat *dThetaPtr) {
    extern __shared__ NFloat tmpPtr[];
    int thdIdx, colIdx, thdNum, base, idx, incr, pos, step, off = THREADPERBLOCK;
    NFloat floatVal, fracVal;

    thdIdx = threadIdx.x;       /* num threads per block */
    colIdx = blockIdx.x;        /* block index, i.e., row num (minibatch size)  */
    thdNum = min(blockDim.x, row);
    if (thdIdx < thdNum && colIdx < col) {
        /*base = colIdx;*/
        /* collect the sums for the groups (and transpose the matrix) */
        tmpPtr[thdIdx] = 0.0;	/*actPtr[base + idx * col];*/
        tmpPtr[off + thdIdx] = 0.0;
        tmpPtr[off + off + thdIdx] = 0.0;
        base = colIdx;
        idx = thdIdx;
        pos = base + idx * col;
        step = thdNum * col;
        while (idx < row) {
            floatVal = (-1.0) * gammaPtr[colIdx] * inpPtr[pos] + thetaPtr[colIdx];
            CHKNFLTEXPE(floatVal)
            fracVal = 1.0 / (1.0 + exp(floatVal));
            tmpPtr[thdIdx] += errPtr[pos] * fracVal;
            if (etaPtr[colIdx] != 0.0) {
                tmpPtr[off + thdIdx] += errPtr[pos] * inpPtr[pos] * etaPtr[colIdx] * fracVal * (1.0 - fracVal);
                tmpPtr[off + off + thdIdx] -= errPtr[pos] * etaPtr[colIdx] * fracVal * (1.0 - fracVal);
            }  
            pos += step;
            idx += thdNum;
        }
        __syncthreads();
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx) {
                    tmpPtr[thdIdx] += tmpPtr[pos];
                    tmpPtr[off + thdIdx] += tmpPtr[off + pos];
                    tmpPtr[off + off + thdIdx] += tmpPtr[off + off + pos];
                }
            }
            __syncthreads();
        }

        /* copy the value to dstPtr */
        if (thdIdx == 0) {
            if (accFlag == FALSE) {
                dEtaPtr[colIdx] = 0.0;
                dGammaPtr[colIdx] = 0.0;
                dThetaPtr[colIdx] = 0.0;
            }
            dEtaPtr[colIdx] += tmpPtr[0];
            dGammaPtr[colIdx] += tmpPtr[off + 0];
            dThetaPtr[colIdx] += tmpPtr[off + off + 0];
        }
    }
}


__global__ void HKern_ApplyPSigmoidAct(NFloat *srcPtr, int row, int col, NFloat *etaPtr, NFloat *dstPtr) {
    int pos, colIdx;
    NFloat floatVal;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < row * col) {
        colIdx = pos % col;
        floatVal = (-1.0) * srcPtr[pos];
        CHKNFLTEXPE(floatVal)
        dstPtr[pos] = etaPtr[colIdx] / (1.0 + exp(floatVal));
    }
}

__global__ void HKern_ApplyDPSigmoidAct(NFloat *srcPtr, int row, int col, NFloat *etaPtr, NFloat *dstPtr) {
    int pos, colIdx;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < row * col) {
        colIdx = pos % col;
        /* dstPtr[pos] = srcPtr[pos] * (1.0 - srcPtr[pos] / etaPtr[colIdx]); */
        if (etaPtr[colIdx] != 0.0)
            dstPtr[pos] = 1.0 / etaPtr[colIdx] * srcPtr[pos] * (etaPtr[colIdx] - srcPtr[pos]);
        else
            dstPtr[pos] = 0.0;
    }
}

__global__ void HKern_ApplyTrPSigmoidActCUDA(NFloat *errPtr, NFloat *srcPtr, NFloat *etaPtr, int row, int col, Boolean accFlag, NFloat *dEtaPtr) {
    extern __shared__ NFloat tmpPtr[];
    int thdIdx, colIdx, thdNum, base, idx, incr, pos, step;

    thdIdx = threadIdx.x;       /* num threads per block */
    colIdx = blockIdx.x;        /* block index, i.e., row num (minibatch size)  */
    thdNum = min(blockDim.x, row);
    if (thdIdx < thdNum && colIdx < col) {
        /*base = colIdx;*/
        /* collect the sums for the groups (and transpose the matrix) */
        tmpPtr[thdIdx] = 0.0;	/*actPtr[base + idx * col];*/
        base = colIdx;
        idx = thdIdx;
        pos = base + idx * col;
        step = thdNum * col;
        while (idx < row) {
            /* tmpPtr[thdIdx] += errPtr[pos] * srcPtr[pos] / etaPtr[colIdx]; */
            if (etaPtr[colIdx] != 0.0)
                tmpPtr[thdIdx] += errPtr[pos] * 1.0 / etaPtr[colIdx] * srcPtr[pos];
            pos += step;
            idx += thdNum;
        }
        __syncthreads();
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx) {
                    tmpPtr[thdIdx] += tmpPtr[pos];
                }
            }
            __syncthreads();
        }

        /* copy the value to dstPtr */
        if (thdIdx == 0) {
            if (accFlag == FALSE)
                dEtaPtr[colIdx] = 0.0;
            dEtaPtr[colIdx] += tmpPtr[0];
        }
    }
}


__global__ void HKern_ApplySigmoidAct(NFloat *srcPtr, int len, NFloat *dstPtr) {
    int pos;
    NFloat floatVal;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < len) {
        floatVal = -1.0 * srcPtr[pos];
        CHKNFLTEXPE(floatVal)
        dstPtr[pos] = 1.0 / (1.0 + exp(floatVal));
    }
}

__global__ void HKern_ApplyDSigmoidAct(NFloat *srcPtr, int len, NFloat *dstPtr) {
    int pos;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < len) {
        dstPtr[pos] = (1 - srcPtr[pos]) * srcPtr[pos];
    }
}

__global__ void HKern_ApplyTanHAct(NFloat *srcPtr, int len, NFloat *dstPtr) {
    int pos;
    NFloat floatVal;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < len) {
        floatVal = srcPtr[pos];
        CHKNFLTEXPE(floatVal)
        floatVal = exp(floatVal);
        dstPtr[pos] = (floatVal - 1.0 / floatVal) / (floatVal + 1.0 / floatVal);
    }
}

__global__ void HKern_ApplyDTanHAct(NFloat *srcPtr, int len, NFloat *dstPtr) {
    int pos;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < len) {
        dstPtr[pos] = 1 - pow(srcPtr[pos], 2);
    }
}

__global__ void HKern_DualSumByRow(NFloat *srcPtr, int col, int size, int incr, NFloat *dstPtr) {
    int lhpos, rhpos, lhidx, rhidx, mod;

    lhpos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (lhpos < size) {
        mod = incr * 2;
        lhidx = lhpos % col;
        if (lhidx % mod == 0) {
            rhidx = lhidx + incr;
            rhpos = lhpos + incr;
            if (rhidx >= col) {
                dstPtr[lhpos] = srcPtr[lhpos];
            }
            else {
                dstPtr[lhpos] = srcPtr[lhpos] + srcPtr[rhpos];
            }
        }
    }
}

__global__ void HKern_ApplySoftmaxAct(NFloat *srcPtr, int row, int col, NFloat *dstPtr) {
    int frame, i, base, off;
    NFloat den, floatVal;

    frame = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (frame < row) {
        den = 0.0;
        base = frame * col;
        for (i = 0, off = base; i < col; ++i, ++off) {
            floatVal = srcPtr[off];
            CHKNFLTEXPE(floatVal)
            floatVal = exp(floatVal);
            dstPtr[off] = floatVal;
            den += floatVal;
        }
        for (i = 0, off = base; i < col; ++i, ++off) {
            dstPtr[off] /= den;
        }
    }
}

__global__ void HKern_ApplyRedSoftmaxAct(NFloat *srcPtr, int row, int col, NFloat *dstPtr) {
    extern __shared__ NFloat tmpPtr[];
    int thdIdx, rowIdx, thdNum, base, idx, incr, pos;
    NFloat maxVal, sumVal, tmpVal;

    thdIdx = threadIdx.x;	/* num threads per block */
    rowIdx = blockIdx.x;	/* block index, i.e., row num (minibatch size)  */
    thdNum = min(blockDim.x, col);
    if (thdIdx < thdNum && rowIdx < row) {
        base = rowIdx * col;
        /* 1. find the max val for current frame (rowIdx) and store it in tmpPtr[thdIdx] */
        /* a. collect the maxes for the groups */
        idx = thdIdx;
        tmpPtr[thdIdx] = srcPtr[base + idx];
        idx += thdNum;
        while (idx < col) {
            pos = base + idx;
            if (tmpPtr[thdIdx] < srcPtr[pos])
                tmpPtr[thdIdx] = srcPtr[pos];
            idx += thdNum;
        }
        __syncthreads();
        /* b. dual max within current block */
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx && tmpPtr[thdIdx] < tmpPtr[pos]) {
                    tmpPtr[thdIdx] = tmpPtr[pos];
                }
            }
            __syncthreads();
        }
        maxVal = tmpPtr[0];
        __syncthreads();
        /* 2. find the sum */
        /* a. collect the sum for the groups */
        idx = thdIdx;
        tmpPtr[thdIdx] = 0.0;
        while (idx < col) {
            pos = base + idx;
            tmpVal = srcPtr[pos] - maxVal;
            CHKNFLTEXPE(tmpVal)
            dstPtr[pos] = exp(tmpVal);
            tmpPtr[thdIdx] += dstPtr[pos];
            idx += thdNum;
        }
        __syncthreads();
        /* b. dual add within current block */
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx) {
                    tmpPtr[thdIdx] += tmpPtr[pos];
                }
            }
            __syncthreads();
        }

        sumVal = tmpPtr[0];
        /* 3. normalise */
        idx = thdIdx; 
        while (idx < col) {
            dstPtr[base + idx] /= sumVal;
            idx += thdNum;
        }
    } 
}

__global__ void HKern_ApplySoftReLAct(NFloat *srcPtr, int len, NFloat *dstPtr) {
    int pos;
    NFloat floatVal;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < len) {
        floatVal = srcPtr[pos];
        CHKNFLTEXPE(floatVal)
        dstPtr[pos] = log(1.0 + exp(floatVal));
    } 
}

__global__ void HKern_ApplyDSoftReLAct(NFloat *srcPtr, int len, NFloat *dstPtr) {
    int pos;
    NFloat floatVal;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < len) {
        floatVal = srcPtr[pos];
        CHKNFLTEXPE(floatVal)
        dstPtr[pos] = 1.0 - 1.0 / exp(floatVal);
    }
}

__global__ void HKern_ApplySoftSignAct(NFloat *srcPtr, int len, NFloat *dstPtr) {
    int pos;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < len) {
        dstPtr[pos] = srcPtr[pos] / (1 + abs(srcPtr[pos]));
    }
}

__global__ void HKern_ApplyLogTrans(NFloat *srcPtr, int len, NFloat *dstPtr) {
    int pos;
    NFloat floatVal;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < len) {
        floatVal = srcPtr[pos];
        if (floatVal <= 0) {
            floatVal = LZERO;
        }
        else {        
            floatVal = log(floatVal);
            if (floatVal < LSMALL) {
                floatVal = LSMALL;
            }
        }
        dstPtr[pos] = floatVal;
    }
}

__global__ void HKern_RedSumNMatrixByColCUDA(NFloat *srcPtr, int row, int col, Boolean accFlag, NFloat *dstPtr) {
    extern __shared__ NFloat tmpPtr[];
    int thdIdx, colIdx, thdNum, base, idx, incr, pos, step;

    thdIdx = threadIdx.x;       /* num threads per block */
    colIdx = blockIdx.x;        /* block index, i.e., row num (minibatch size)  */
    thdNum = min(blockDim.x, row);
    if (thdIdx < thdNum && colIdx < col) {
        /*base = colIdx;*/
        /* collect the sums for the groups (and transpose the matrix) */
        tmpPtr[thdIdx] = 0.0;/*srcPtr[base + idx * col];*/
        base = colIdx;
        idx = thdIdx;
        pos = base + idx * col;
        step = thdNum * col;
        while (idx < row) {
            tmpPtr[thdIdx] += srcPtr[pos];
            pos += step;
            idx += thdNum;
        }
        __syncthreads();
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx) {
                    tmpPtr[thdIdx] += tmpPtr[pos];
                }
            }
            __syncthreads();
        }

        /* copy the value to dstPtr */
        if (thdIdx == 0) {
            if (accFlag == FALSE) 
                dstPtr[colIdx] = 0.0; 
            dstPtr[colIdx] += tmpPtr[0];
        }
    }
}

__global__ void HKern_SumNMatrixByCol(NFloat *srcPtr, int row, int col, NFloat *dstPtr) {
    int i, pos;
    NFloat sum;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < col) {
        sum = 0.0;
        for (i = 0; i < row; ++i) {
            sum += srcPtr[i * col + pos];
        }
        dstPtr[pos] = sum;
    }
}

__global__ void HKern_SumNMatrixByColAcc(NFloat *srcPtr, int row, int col, NFloat *dstPtr) {
    int i, pos;
    NFloat sum;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < col) {
        sum = 0.0;
        for (i = 0; i < row; ++i) {
            sum += srcPtr[i * col + pos];
        }
        dstPtr[pos] += sum;
    }
}

__global__ void HKern_SquaredNSegment(NFloat *srcPtr, int segLen, NFloat *dstPtr) {
    int pos;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < segLen) {
        dstPtr[pos] = pow(srcPtr[pos], 2);
    }
}

__global__ void HKern_CompAdaGradNSegment(NFloat eta, int K, int segLen, NFloat *ssgSeg, NFloat *nlrSeg) {
    int pos;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < segLen) {
        nlrSeg[pos] = eta / sqrt(K + ssgSeg[pos]);
    }
}

__global__ void HKern_CalXENTCriterionCUDA(NFloat *refPtr, NFloat *hypPtr, int segLen, NFloat *crtPtr) {
    __shared__ NFloat tmpPtr[THREADPERBLOCK];
    int thdIdx, thdNum, pos, idx, incr;
    NFloat tn, yn;

    thdIdx = threadIdx.x;
    thdNum = blockDim.x;

    if (thdIdx < thdNum) {
        /* a. collect the sums for the groups */
        pos = thdIdx;
        tmpPtr[thdIdx] = 0.0;
        while (pos < segLen) {
            tn = refPtr[pos];
            yn = hypPtr[pos];
            if (tn == 0.0) 
                tmpPtr[thdIdx] += 0.0;
            else if (yn == 0.0) 
                tmpPtr[thdIdx] += tn * LZERO;
            else 
                tmpPtr[thdIdx] += (-1.0) * tn * log(yn / tn); 
            pos += thdNum;
        }
        __syncthreads();
        /* b. dual add within current block */
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx) 
                    tmpPtr[thdIdx] += tmpPtr[pos];
            }
            __syncthreads();
        }
        *crtPtr = tmpPtr[0];
    } 
}

__global__ void HKern_CalMMSECriterionCUDA(NFloat *refPtr, NFloat *hypPtr, int segLen, NFloat *crtPtr) {
    __shared__ NFloat tmpPtr[THREADPERBLOCK];
    int thdIdx, thdNum, pos, idx, incr;

    thdIdx = threadIdx.x;
    thdNum = blockDim.x;
    
    if (thdIdx < thdNum) {
        /* a. collect the sums for the groups */
        pos = thdIdx;
        tmpPtr[thdIdx] = 0.0;
        while (pos < segLen) {
            tmpPtr[thdIdx] += pow(refPtr[pos] - hypPtr[pos], 2);
            pos += thdNum;
        }
        __syncthreads();
        /* dual add within current block */
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx) {
                    tmpPtr[thdIdx] += tmpPtr[pos];
                }
            }
            __syncthreads();
        }
        *crtPtr = tmpPtr[0];
    }
}

__global__ void HKern_AddSegmentTargetPen(NFloat *srcPtr, NFloat *penPtr, int row, int col, NFloat *dstPtr) {
    int pos, off;
    
    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < row * col) {
        off = pos % col;
        dstPtr[pos] = srcPtr[pos] + penPtr[off];
    }
}

/*__global__ void HKern_SubNSegmentByConst(NFloat *srcSeg, int segLen, float constVal, NFloat *dstSeg) {
    int pos;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < segLen) {
        dstSeg[pos] = srcSeg[pos] - constVal;
    }
}*/

/* cz277 - semi */
__global__ void HKern_ShiftNSegmentVals(NFloat *srcSeg, int segLen, float shiftVal, NFloat *dstSeg) {
    int pos;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < segLen) {
        dstSeg[pos] = srcSeg[pos] + shiftVal;
    }
}

/* cz277 - 1007 */
__global__ void HKern_CopyPartialNSegment(int minRow, int minCol, NFloat *srcPtr, int srcCol, NFloat *dstPtr, int dstCol) {
    int pos, rowIdx, colIdx;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < minRow * minCol) {
        rowIdx = pos / minCol;
        colIdx = pos % minCol;
        dstPtr[rowIdx * dstCol + colIdx] = srcPtr[rowIdx * srcCol + colIdx];
    }
}

/* cz277 - gradlim */
__global__ void HKern_ClipNSegmentVals(NFloat* srcSeg, int len, NFloat upperLim, NFloat lowerLim, NFloat *dstSeg) {
    int pos;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < len) {
        if (srcSeg[pos] > upperLim)
            dstSeg[pos] = upperLim;
        else if (srcSeg[pos] < lowerLim)
            dstSeg[pos] = lowerLim;
        else if (srcSeg != dstSeg)
            dstSeg[pos] = srcSeg[pos];
    }
}

__global__ void HKern_RedMaxElementIndex(NFloat *srcPtr, int row, int col, NFloat *dstPtr) {
    extern __shared__ NFloat tmpPtr[];
    int thdIdx, rowIdx, thdNum, base, idx, incr, pos, off = THREADPERBLOCK;

    thdIdx = threadIdx.x;       /* num threads per block */
    rowIdx = blockIdx.x;        /* block index, i.e., row num (minibatch size)  */
    thdNum = min(blockDim.x, col);
    if (thdIdx < thdNum && rowIdx < row) {
        base = rowIdx * col;
        /* find the max val for current frame (rowIdx) and store it in tmpPtr[thdIdx] */
        /* a. collect the maxes for the groups */
        idx = thdIdx;
        tmpPtr[thdIdx] = srcPtr[base + idx];
        tmpPtr[off + thdIdx] = idx;
        idx += thdNum;
        while (idx < col) {
            pos = base + idx;
            if (tmpPtr[thdIdx] < srcPtr[pos]) {
                tmpPtr[thdIdx] = srcPtr[pos];
                tmpPtr[off + thdIdx] = idx;
            }
            idx += thdNum;
        }
        __syncthreads();
        /* b. dual max within current block */
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx && tmpPtr[thdIdx] < tmpPtr[pos]) {
                    tmpPtr[thdIdx] = tmpPtr[pos];
                    tmpPtr[off + thdIdx] = tmpPtr[off + pos];
                }
            }
            __syncthreads();
        }
        /*__syncthreads();*/
        if (thdIdx == 0)
            dstPtr[rowIdx] = tmpPtr[off + 0];
            /*dstPtr[rowIdx] = (NFloat) tmpPtr[off + 0];*/
        /*__syncthreads();*/
    }	
}

/* cz277 - max norm */
__global__ void HKern_RedCalNMatrixL2NormByRow(NFloat *matPtr, int row, int col, NFloat *normPtr) {
    extern __shared__ NFloat tmpPtr[];
    int thdIdx, rowIdx, thdNum, base, idx, incr, pos;

    thdIdx = threadIdx.x;       /* num threads per block */
    rowIdx = blockIdx.x;        /* block index, i.e., row num (minibatch size)  */
    thdNum = min(blockDim.x, col);
    if (thdIdx < thdNum && rowIdx < row) {
        /* 1. accumulate the L2 norm for each row */
        base = rowIdx * col;
        idx = thdIdx;
        tmpPtr[thdIdx] = 0.0;
        while (idx < col) {
            pos = base + idx;
            tmpPtr[thdIdx] += pow(matPtr[pos], 2);
            idx += thdNum;
        }
        __syncthreads();
        /* b. dual add within current block */
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++incr;                                   
	    }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx) {
                    tmpPtr[thdIdx] += tmpPtr[pos];
                }
            }
            __syncthreads();
        }
        if (thdIdx == 0) {
            normPtr[rowIdx] = sqrt(tmpPtr[0]);
        }
    }
}

/* cz277 - max norm */
__global__ void HKern_RedMaxElementValue(NFloat *srcPtr, int row, int col, NFloat *dstPtr) {
    extern __shared__ NFloat tmpPtr[];
    int thdIdx, rowIdx, thdNum, base, idx, incr, pos;

    thdIdx = threadIdx.x;       /* num threads per block */
    rowIdx = blockIdx.x;        /* block index, i.e., row num (minibatch size)  */
    thdNum = min(blockDim.x, col);
    if (thdIdx < thdNum && rowIdx < row) {
        base = rowIdx * col;
        /* find the max val for current frame (rowIdx) and store it in tmpPtr[thdIdx] */
        /* a. collect the maxes for the groups */
        idx = thdIdx;
        tmpPtr[thdIdx] = srcPtr[base + idx];
        idx += thdNum;
        while (idx < col) {
            pos = base + idx;
            if (tmpPtr[thdIdx] < srcPtr[pos]) {
                tmpPtr[thdIdx] = srcPtr[pos];
            }
            idx += thdNum;
        }
        __syncthreads();
        /* b. dual max within current block */
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx && tmpPtr[thdIdx] < tmpPtr[pos]) {
                    tmpPtr[thdIdx] = tmpPtr[pos];
                }
            }
            __syncthreads();
        }
        /*__syncthreads();*/
        if (thdIdx == 0)
            dstPtr[rowIdx] = sqrt(tmpPtr[0]);
    }
}

__global__ void HKern_DivideNMatrixByRow(NFloat *srcPtr, int row, int col, NFloat *normPtr, NFloat *dstPtr) {
    int pos, rowIdx;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < row * col) {
        rowIdx = (int) pos / col;
        dstPtr[pos] = srcPtr[pos] / normPtr[rowIdx];
    }
}

/* --------------------------- HFBLat Kerns ------------------------ */

/* cz277 - cuda fblat */
__global__ void HKern_Setotprob4q(int T, NFloat *llhPtr, int ncols, int *qLo, int *qHi, int Q, float probScale, AcousticDev *acList) {
    int pos, tIdx, tRel, qIdx, s, Nq1;
    AcousticDev *curAc;
    NFloat *otprob;
    NFloat *matptr;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < T * Q) {
        tIdx = pos / Q + 1;
        qIdx = pos % Q + 1;
        if (qIdx >= qLo[tIdx] && qIdx <= qHi[tIdx]) {
            curAc = &acList[qIdx];
            Nq1 = curAc->Nq + 1;
            if (tIdx >= curAc->t_start && tIdx <= curAc->t_end) {	/* q is active at t */
                matptr = llhPtr + (tIdx - 1) * ncols;
                tRel = tIdx - curAc->t_start + 1;
                otprob = curAc->otprob + tRel * Nq1;
                for (s = 2; s < curAc->Nq; ++s) {
                    otprob[s] = matptr[curAc->indexes[s] - 1];
                }
            }
        }
    }
}


/* cz277 - cuda fblat */
__device__ NFloat LAddDev(NFloat x, NFloat y) {
    NFloat temp, diff, z;

    if (x < y) {
        temp = x;
        x = y;
        y = temp;
    }
    diff = y - x;
    if (diff < -23.025851) {
        if (x < LSMALL) {
            return LZERO;
        }
        else {
            return x;
        }
    }
    else {
        z = exp(diff);
        return x + log(1.0 + z);
    }
}

/* cz277 - cuda fblat */
__global__ void HKern_SetModelPlus(int Q, AcousticDev *acList) {
    int tIdx, tRel, qIdx, Nq1, i, j;
    AcousticDev *curAc;
    NFloat *bqt, *bqt1, x;

    qIdx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (qIdx < Q) {
        qIdx += 1;
        curAc = acList + qIdx;
        Nq1 = curAc->Nq + 1;
        for (tIdx = curAc->t_end; tIdx >= curAc->t_start; --tIdx) {
            tRel = tIdx - curAc->t_start + 1;
            /* SetModelPlus subroutine */
            x = LZERO;
            bqt = &curAc->betaPlus[tRel * Nq1];
            bqt1 = &curAc->betaPlus[(tRel + 1) * Nq1];
            if (tIdx == curAc->t_end) 
                bqt[curAc->Nq] = 0;
            else 
                bqt[curAc->Nq] = LZERO;
            for (i = 2; i < curAc->Nq; ++i) {
                x = bqt[curAc->Nq] + curAc->transp[i * Nq1 + curAc->Nq]; 
                if (tIdx + 1 <= curAc->t_end) {	/* in beam next time frame */
                    for (j = 2; j < curAc->Nq; ++j) {
                        x = LAddDev(x, bqt1[j] + curAc->transp[i * Nq1 + j]);
                    }
                }
                x += curAc->otprob[tRel * Nq1 + i];
                bqt[i] = x;
            }
            x = LZERO;
            for (i = 2; i < curAc->Nq; ++i) {
                x = LAddDev(x, bqt[i] + curAc->transp[1 * Nq1 + i]);
            }
            bqt[1] = x;
        }
        /* neet to set the total accumulated acoustics (tRel ~ tIdx = curAc->t_start) */
        if (curAc->SP == TRUE)
            curAc->aclike = curAc->transp[1 * Nq1 + curAc->Nq];
        else
            curAc->aclike = curAc->betaPlus[tRel * Nq1 + 1];
    }
}


/* cz277 - cuda fblat */
__global__ void HKern_ZeroAlphas(int T, int Q, AcousticDev *acList) {
    int i, pos, Nq1, tIdx, tRel, qIdx;
    AcousticDev *curAc;
    NFloat *alpha;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < T * Q) {
        tIdx = pos / Q + 1;
        qIdx = pos % Q + 1;
        curAc = &acList[qIdx];
        /* q is active at t */
        if (tIdx >= curAc->t_start && tIdx <= curAc->t_end) { 
            tRel = tIdx - curAc->t_start + 1;
            Nq1 = curAc->Nq + 1;
            alpha = &curAc->alphaPlus[tRel * Nq1];
            if (curAc->SP == FALSE) {
                for (i = 1; i < Nq1; ++i) {
                    alpha[i] = LZERO;    
                }
            }
        }
    }
}


/* cz277 - cuda fblat */
__global__ void HKern_StepAlpha(int Q, AcousticDev *acList) {
    int tIdx, qIdx, Nq1, i, j, tRel;
    AcousticDev *curAc;
    NFloat *aq, *laq, x = 0.0, y, a;
    NFloat *outprob;

    qIdx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (qIdx < Q) {
        qIdx += 1;
        curAc = acList + qIdx;
        /* for each time */
        for (tIdx = curAc->t_start; tIdx <= curAc->t_end; ++tIdx) {
            tRel = tIdx - curAc->t_start + 1;
            Nq1 = curAc->Nq + 1;
            aq = &curAc->alphaPlus[tRel * Nq1];
            laq = (tIdx - 1 >= curAc->t_start && tIdx - 1 <= curAc->t_end)? &curAc->alphaPlus[(tRel - 1) * Nq1]: NULL;
            /* outprob != NULL ?? */
            outprob = &curAc->otprob[tRel * Nq1];
            if (tIdx == curAc->t_start) 
                aq[1] = curAc->locc - curAc->aclike;
            else 
                aq[1] = LZERO;
            x = LZERO;
            for (j = 2; j < curAc->Nq; ++j) {
                a = curAc->transp[1 * Nq1 + j];
                x = (a > LSMALL)? a + aq[1]: LZERO;
                for (i = 2; i <= curAc->Nq; ++i) {
                    a = curAc->transp[i * Nq1 + j];
                    y = (laq? laq[i]: LZERO);
                    if (a > LSMALL && y > LSMALL) {
                        x = LAddDev(x, y + a);
                        /*x = log(x + y + a);*/
                    }
                }
                aq[j] = x + outprob[j];
            }
            x = LZERO;
            for (i = 2; i < curAc->Nq; ++i) {
                a = curAc->transp[i * Nq1 + curAc->Nq];
                y = aq[i];
                if (a > LSMALL && y > LSMALL) {
                    x = LAddDev(x, y + a);
                    /*x = log(x + y + a);*/
                }
            }
	    aq[curAc->Nq] = x;
            /* work out the exit problem for checking purpose */
        }
    }
}


/* --------------------------- Trace Flags ------------------------ */

/*  */
void SyncDev2Host(void *devPtr, void *hostPtr, size_t size) {
    hipMemcpy(hostPtr, devPtr, size, hipMemcpyDeviceToHost);
}

/*  */
void SyncHost2Dev(void *hostPtr, void *devPtr, size_t size) {
    hipMemcpy(devPtr, hostPtr, size, hipMemcpyHostToDevice);	
}

/*  */
void DevDispose(void *devPtr, size_t size) {
    hipFree(devPtr);
    GPUMemUsed -= size;
}

/*  */
Boolean DevNew(void **devAddr, size_t size) {
    if (hipMalloc(devAddr, size) != hipSuccess)
        return FALSE;
    GPUMemUsed += size;
    return TRUE;
}

/*  */
void ShowGPUMemUsage(void) {
    printf("(More than) %luMB space allocated in GPU %d memory\n", GPUMemUsed / 1048576, GPUDevId);
}

/*  */
void SetNSegmentCUDA(NFloat val, NFloat *segPtr, int segLen) {
    int nBlocks;

    nBlocks = CEIL(segLen, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"SetNSegmentCUDA: Block number exceeds the maximum");
    HKern_SetNSegment<<<nBlocks, THREADPERBLOCK>>>(val, segPtr, segLen);
}

/*  */
void ClearNSegmentCUDA(NFloat *segPtr, int segLen) {
    int nBlocks;
    hipError_t status;

    nBlocks = CEIL(segLen, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"ClearNSegmentCUDA: Block number exceeds the maximum");
    /*HKern_SetNSegment<<<nBlocks, THREADPERBLOCK>>>(0, segPtr, segLen);*/
    status = hipMemset(segPtr, 0, segLen * sizeof(NFloat));
    if (status != hipSuccess) 
        HError(8822, (char *)"ClearNSegmentCUDA: hipMemset funtion failed");
    /*hipDeviceSynchronize();*/
}


/*  */
void CopyNSegmentCUDA(NFloat *srcPtr, int segLen, NFloat *dstPtr) {
    hipblasStatus_t status;

#ifdef DOUBLEANN
    status = hipblasDcopy(handle, segLen, srcPtr, 1, dstPtr, 1);
#else
    status = hipblasScopy(handle, segLen, srcPtr, 1, dstPtr, 1);
#endif
    if (status != HIPBLAS_STATUS_SUCCESS) 
        HError(8822, (char *)"CopyNSegmentCUDA: CUBLAS library copy function failed");
}

/*  */
void AddNSegmentCUDA(NFloat *srcPtr, int segLen, NFloat *dstPtr) {
    hipblasStatus_t status;
    const NFloat alpha = 1.0;

#ifdef DOUBLEANN
    status = hipblasDaxpy(handle, segLen, &alpha, srcPtr, 1, dstPtr, 1);
#else
    status = hipblasSaxpy(handle, segLen, &alpha, srcPtr, 1, dstPtr, 1);
#endif

    if (status != HIPBLAS_STATUS_SUCCESS) 
        HError(8822, (char *)"AddNSegmentCUDA: CUBLAS library copy function failed");
}

/* cz277 - l2 fix */
void AddScaledNSegmentCUDA(NFloat *srcPtr, int segLen, NFloat scale, NFloat *dstPtr) {
    hipblasStatus_t status;
    const NFloat alpha = scale;

#ifdef DOUBLEANN
    status = hipblasDaxpy(handle, segLen, &alpha, srcPtr, 1, dstPtr, 1);
#else
    status = hipblasSaxpy(handle, segLen, &alpha, srcPtr, 1, dstPtr, 1);
#endif
    if (status != HIPBLAS_STATUS_SUCCESS) 
        HError(8822, (char *)"AddScaledNSegmentCUDA: CUBLAS library copy function failed");
}

/*  */
void ScaleNSegmentCUDA(int segLen, NFloat scale, NFloat *valPtr) {
    hipblasStatus_t status;

#ifdef DOUBLEANN
    status = hipblasDscal(handle, segLen, &scale, valPtr, 1);
#else
    status = hipblasSscal(handle, segLen, &scale, valPtr, 1);
#endif
    if (status != HIPBLAS_STATUS_SUCCESS) 
        HError(8822, (char *)"ScaleNSegmentCUDA: CUBLAS library copy function failed");
}

/*  */
void ScaledSelfAddNSegmentCUDA(NFloat *rhPtr, int segLen, NFloat scale, NFloat *lhPtr) {
    int nBlocks;
    
    nBlocks = CEIL(segLen, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"ScaledSelfAddNSegmentCUDA: Block number exceeds the maximum");
    HKern_ScaledSelfAddNSegment<<<nBlocks, THREADPERBLOCK>>>(rhPtr, segLen, scale, lhPtr);
}

/*  */
void DupNSegmentCUDA(NFloat *srcPtr, int segLen, NFloat *dstPtr, int times) {
    int nBlocks;

    nBlocks = CEIL(segLen * times, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"DupNSegmentCUDA: Block number exceeds the maximum");
    HKern_DupNSegment<<<nBlocks, THREADPERBLOCK>>>(srcPtr, segLen, dstPtr, times);
}

/*  */
void SubNSegmentCUDA(NFloat *lhPtr, NFloat *rhPtr, int segLen, NFloat *resPtr) {
    int nBlocks;
  
    nBlocks = CEIL(segLen, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"SubNSegmentCUDA: Block number exceeds the maximum");
    HKern_SubNSegment<<<nBlocks, THREADPERBLOCK>>>(lhPtr, rhPtr, segLen, resPtr);
}

/*  */
void MulNSegmentCUDA(NFloat *lhPtr, NFloat *rhPtr, int segLen, NFloat *resPtr) {
    int nBlocks;

    nBlocks = CEIL(segLen, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"MulNSegmentCUDA: Block number exceeds the maximum");
    HKern_MulNSegment<<<nBlocks, THREADPERBLOCK>>>(lhPtr, rhPtr, segLen, resPtr);
}

/* cz277 - pact */
void ApplyAffineActCUDA(NFloat *srcPtr, int row, int col, NFloat *scalePtr, NFloat *shiftPtr, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(row * col, THREADPERBLOCK);    
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"ApplyAffineActCUDA: Block number exceeds the maximum");
    HKern_ApplyAffineAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, row, col, scalePtr, shiftPtr, dstPtr);
}

/* cz277 - pact */
void ApplyDAffineActCUDA(NFloat *srcPtr, int row, int col, NFloat *scalePtr, NFloat *shiftPtr, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(row * col, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"ApplyDAffineActCUDA: Block number exceeds the maximum");
    HKern_ApplyDAffineAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, row, col, scalePtr, shiftPtr, dstPtr);
}


/* cz277 - pact */
void ApplyTrAffineActCUDA(NFloat *errPtr, NFloat *actPtr, int row, int col, NFloat *scalePtr, NFloat *shiftPtr, Boolean accFlag, NFloat *dScalePtr, NFloat *dShiftPtr) {
    int nBlocks, sBytes;

    nBlocks = col;
    sBytes = 2 * sizeof(NFloat) * THREADPERBLOCK;
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"ApplyTrStdDevAffineActCUDA: Block number exceeds the maximum");
    HKern_ApplyTrAffineAct<<<nBlocks, THREADPERBLOCK, sBytes>>>(errPtr, actPtr, row, col, scalePtr, shiftPtr, accFlag, dScalePtr, dShiftPtr);
}


/* cz277 - laf */
void AccMeanNSegmentCUDA(NFloat *valPtr, int row, int col, NFloat tSamp, NFloat *meanPtr) {
    int nBlocks, sBytes;
    
    nBlocks = col;
    sBytes = sizeof(NFloat) * THREADPERBLOCK;
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"AccMeanNSegmentCUDA: Block number exceeds the maximum");
    HKern_AccMeanNSegment<<<nBlocks, THREADPERBLOCK, sBytes>>>(valPtr, row, col, tSamp, meanPtr);
}

/* cz277 - laf */
void AccVarianceNSegmentCUDA(NFloat *valPtr, int row, int col, NFloat tSamp, NFloat *meanPtr, NFloat *varPtr) {
    int nBlocks, sBytes;
    
    nBlocks = col;
    sBytes = sizeof(NFloat) * THREADPERBLOCK;
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"AccVarianceNSegmentCUDA: Block number exceeds the maximum");
    HKern_AccVarianceNSegment<<<nBlocks, THREADPERBLOCK, sBytes>>>(valPtr, row, col, tSamp, meanPtr, varPtr);
}

/* cz277 - pact */
void ApplyParmReLUActCUDA(NFloat *srcPtr, int row, int col, NFloat *posPtr, NFloat *negPtr, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(row * col, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"ApplyParmReLUActCUDA: Block number exceeds the maximum");
    HKern_ApplyParmReLUAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, row, col, posPtr, negPtr, dstPtr);
}

/* cz277 - pact */
void ApplyDParmReLUActCUDA(NFloat *inpPtr, int row, int col, NFloat *posPtr, NFloat *negPtr, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(row * col, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"ApplyDParmReLUActCUDA: Block number exceeds the maximum");
    HKern_ApplyDParmReLUAct<<<nBlocks, THREADPERBLOCK>>>(inpPtr, row, col, posPtr, negPtr, dstPtr);
}

/* cz277 - pact */
void ApplyTrParmReLUActCUDA(NFloat *errPtr, NFloat *inpPtr, int row, int col, Boolean accFlag, NFloat *dPosPtr, NFloat *dNegPtr) {
    int nBlocks, sBytes;

    nBlocks = col;
    sBytes = 2 * sizeof(NFloat) * THREADPERBLOCK;
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"ApplyTrParmReLUActCUDA: Block number exceeds the maximum");
    HKern_ApplyTrParmReLUAct<<<nBlocks, THREADPERBLOCK, sBytes>>>(errPtr, inpPtr, row, col, accFlag, dPosPtr, dNegPtr);
}

/* cz277 - pact */
void ApplyPReLUActCUDA(NFloat *srcPtr, int row, int col, NFloat *scalePtr, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(row * col, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"ApplyPReLUActCUDA: Block number exceeds the maximum");
    HKern_ApplyPReLUAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, row, col, scalePtr, dstPtr);
}

/* cz277 - pact */
void ApplyDPReLUActCUDA(NFloat *srcPtr, int row, int col, NFloat *scalePtr, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(row * col, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"ApplyDPReLUActCUDA: Block number exceeds the maximum");
    HKern_ApplyDPReLUAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, row, col, scalePtr, dstPtr);
}

/* cz277 - pact */
void ApplyTrPReLUActCUDA(NFloat *errPtr, NFloat *srcPtr, int row, int col, NFloat *scalePtr, Boolean accFlag, NFloat *dScalePtr) {
    int nBlocks, sBytes;

    nBlocks = col;
    sBytes = sizeof(NFloat) * THREADPERBLOCK;
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"ApplyTrPReLUActCUDA: Block number exceeds the maximum");
    HKern_ApplyTrPReLUAct<<<nBlocks, THREADPERBLOCK, sBytes>>>(errPtr, srcPtr, row, col, scalePtr, accFlag, dScalePtr);
}

/*  */
void ApplyReLUActCUDA(NFloat *srcPtr, int len, NFloat scale, NFloat *dstPtr) {
    int nBlocks;
    
    nBlocks = CEIL(len, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"ApplyReLActCUDA: Block number exceeds the maximum");
    HKern_ApplyReLUAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, len, scale, dstPtr);
}

/*  */
void ApplyDReLUActCUDA(NFloat *srcPtr, int len, NFloat scale, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(len, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"ApplyDReLActCUDA: Block number exceeds the maximum");
    HKern_ApplyDReLUAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, len, scale, dstPtr);
}

/*  */
void ApplyDLinearActCUDA(NFloat *srcPtr, int len, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(len, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"ApplyDLinearActCUDA: Block number exceeds the maximum");
    HKern_ApplyDLinearAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, len, dstPtr);
}

void ApplyLHUCSigmoidActCUDA(NFloat *srcPtr, int row, int col, NFloat *rolePtr, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(row * col, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"ApplyLHUCSigmoidActCUDA: Block number exceeds the maximum");
    HKern_ApplyLHUCSigmoidAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, row, col, rolePtr, dstPtr);
}

void ApplyDLHUCSigmoidActCUDA(NFloat *srcPtr, int row, int col, NFloat *rolePtr, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(row * col, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"ApplyDLHUCSigmoidActCUDA: Block number exceeds the maximum");
    HKern_ApplyDLHUCSigmoidAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, row, col, rolePtr, dstPtr);
}

void ApplyTrLHUCSigmoidActCUDA(NFloat *errPtr, NFloat *actPtr, int row, int col, NFloat *rolePtr, Boolean accFlag, NFloat *dRolePtr) {
    int nBlocks, sBytes;

    nBlocks = col;
    sBytes = sizeof(NFloat) * THREADPERBLOCK;
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"ApplyTrLHUCSigmoidActCUDA: Block number exceeds the maximum");
    HKern_ApplyTrLHUCSigmoidActCUDA<<<nBlocks, THREADPERBLOCK, sBytes>>>(errPtr, actPtr, row, col, rolePtr, accFlag, dRolePtr); 
}

void ApplyPSigmoidActCUDA(NFloat *srcPtr, int row, int col, NFloat *etaPtr, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(row * col, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"ApplyPSigmoidActCUDA: Block number exceeds the maximum");
    HKern_ApplyPSigmoidAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, row, col, etaPtr, dstPtr);
}

void ApplyDPSigmoidActCUDA(NFloat *srcPtr, int row, int col, NFloat *etaPtr, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(row * col, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"ApplyDPSigmoidActCUDA: Block number exceeds the maximum");
    HKern_ApplyDPSigmoidAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, row, col, etaPtr, dstPtr);
}

void ApplyTrPSigmoidActCUDA(NFloat *errPtr, NFloat *srcPtr, NFloat *etaPtr, int row, int col, Boolean accFlag, NFloat *dEtaPtr) {
    int nBlocks, sBytes;

    nBlocks = col;
    sBytes = sizeof(NFloat) * THREADPERBLOCK;
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"ApplyTrPSigmoidActCUDA: Block number exceeds the maximum");
    HKern_ApplyTrPSigmoidActCUDA<<<nBlocks, THREADPERBLOCK, sBytes>>>(errPtr, srcPtr, etaPtr, row, col, accFlag, dEtaPtr);
}


void ApplyParmSigmoidActCUDA(NFloat *srcPtr, int row, int col, NFloat *etaPtr, NFloat *gammaPtr, NFloat *thetaPtr, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(row * col, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"ApplyParmSigmoidActCUDA: Block number exceeds the maximum");
    HKern_ApplyParmSigmoidAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, row, col, etaPtr, gammaPtr, thetaPtr, dstPtr);
}

void ApplyDParmSigmoidActCUDA(NFloat *srcPtr, int row, int col, NFloat *etaPtr, NFloat *gammaPtr, NFloat *thetaPtr, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(row * col, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"ApplyDParmSigmoidActCUDA: Block number exceeds the maximum");
    HKern_ApplyDParmSigmoidAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, row, col, etaPtr, gammaPtr, thetaPtr, dstPtr);
}

void ApplyTrParmSigmoidActCUDA(NFloat *errPtr, NFloat *inpPtr, int row, int col, NFloat *etaPtr, NFloat *gammaPtr, NFloat *thetaPtr, Boolean accFlag, NFloat *dEtaPtr, NFloat *dGammaPtr, NFloat *dThetaPtr) {
    int nBlocks, sBytes;

    nBlocks = col;
    sBytes = 3 * sizeof(NFloat) * THREADPERBLOCK;
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"ApplyTrParmSigmoidActCUDA: Block number exceeds the maximum");
    HKern_ApplyTrParmSigmoidActCUDA<<<nBlocks, THREADPERBLOCK, sBytes>>>(errPtr, inpPtr, row, col, etaPtr, gammaPtr, thetaPtr, accFlag, dEtaPtr, dGammaPtr, dThetaPtr);
}


/*  */
void ApplySigmoidActCUDA(NFloat *srcPtr, int len, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(len, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"ApplySigmoidActCUDA: Block number exceeds the maximum");
    HKern_ApplySigmoidAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, len, dstPtr);
}

/*  */
void ApplyDSigmoidActCUDA(NFloat *srcPtr, int len, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(len, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"ApplyDSigmoidActCUDA: Block number exceeds the maximum");
    HKern_ApplyDSigmoidAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, len, dstPtr);
}

/*  */
void ApplyTanHActCUDA(NFloat *srcPtr, int len, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(len, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"ApplyTanHActCUDA: Block number exceeds the maximum");
    HKern_ApplyTanHAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, len, dstPtr);
}

/*  */
void ApplyDTanHActCUDA(NFloat *srcPtr, int len, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(len, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"ApplyDTanHActCUDA: Block number exceeds the maximum");
    HKern_ApplyDTanHAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, len, dstPtr);
}


/*  */
void ApplyRedSoftmaxActCUDA(NFloat *srcPtr, int row, int col, NFloat *dstPtr) {
    int nBlocks, sBytes;

    nBlocks = row;
    sBytes = sizeof(NFloat) * THREADPERBLOCK;
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"ApplyRedSoftmaxActCUDA: Block number exceeds the maximum");
    HKern_ApplyRedSoftmaxAct<<<nBlocks, THREADPERBLOCK, sBytes>>>(srcPtr, row, col, dstPtr);
}

/*  */
void ApplySoftmaxActCUDA(NFloat *srcPtr, int row, int col, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(row, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"ApplySoftmaxActCUDA: Block number exceeds the maximum");
    HKern_ApplySoftmaxAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, row, col, dstPtr);
}

/*  */
void ApplySoftReLActCUDA(NFloat *srcPtr, int len, NFloat *dstPtr) {
    int nBlocks;
 
    nBlocks = CEIL(len, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"ApplySoftReLActCUDA: Block number exceeds the maximum");
    HKern_ApplySoftReLAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, len, dstPtr);
}

/*  */
void ApplyDSoftReLActCUDA(NFloat *srcPtr, int len, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(len, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"ApplySoftReLActCUDA: Block number exceeds the maximum");
    HKern_ApplyDSoftReLAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, len, dstPtr);
}

/*  */
void ApplySoftSignActCUDA(NFloat *srcPtr, int len, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(len, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"ApplySoftSignActCUDA: Block number exceeds the maximum");
    HKern_ApplySoftSignAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, len, dstPtr);    
}

/*  */
void ApplyLogTransCUDA(NFloat *srcPtr, int len, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(len, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"ApplyLogTransCUDA: Block number exceeds the maximum");
    HKern_ApplyLogTrans<<<nBlocks, THREADPERBLOCK>>>(srcPtr, len, dstPtr);    
}

/*  */
void RedSumNMatrixByColCUDA(NFloat *srcPtr, int row, int col, Boolean accFlag, NFloat *dstPtr) {
    int nBlocks, sBytes;

    nBlocks = col;
    sBytes = sizeof(NFloat) * THREADPERBLOCK;
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"RedSumNMatrixByColCUDA: Block number exceeds the maximum");
    HKern_RedSumNMatrixByColCUDA<<<nBlocks, THREADPERBLOCK, sBytes>>>(srcPtr, row, col, accFlag, dstPtr);
}

/*  */
void SumNMatrixByColCUDA(NFloat *srcPtr, int row, int col, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(col, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"SumNMatrixByColCUDA: Block number exceeds the maximum");
    HKern_SumNMatrixByCol<<<nBlocks, THREADPERBLOCK>>>(srcPtr, row, col, dstPtr);
}

/*  */
void SquaredNSegmentCUDA(NFloat *srcPtr, int segLen, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(segLen, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"SquaredNSegmentCUDA: Block number exceeds the maximum");
    HKern_SquaredNSegment<<<nBlocks, THREADPERBLOCK>>>(srcPtr, segLen, dstPtr);
}

/*  */
void CompAdaGradNSegmentCUDA(NFloat eta, int K, int segLen, NFloat *ssgSeg, NFloat *nlrSeg) {
    int nBlocks;

    nBlocks = CEIL(segLen, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"CompAdaGradNSegmentCUDA: Block number exceeds the maximum");
    HKern_CompAdaGradNSegment<<<nBlocks, THREADPERBLOCK>>>(eta, K, segLen, ssgSeg, nlrSeg);
}

/*  */
void HNBlasNNgemmCUDA(int m, int n, int k, NFloat alpha, NFloat *A, NFloat *B, NFloat beta, NFloat *C) {
    hipblasStatus_t status;

#ifdef DOUBLEANN
    status = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, A, m, B, k, &beta, C, m);
#else
    status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, A, m, B, k, &beta, C, m);
#endif
    if (status != HIPBLAS_STATUS_SUCCESS) {
        HError(8890, (char *)"HNBlasNNgemmCUDA: CUBLAS library gemm function failed");
    }
}

/*  */
void HNBlasNTgemmCUDA(int m, int n, int k, NFloat alpha, NFloat *A, NFloat *B, NFloat beta, NFloat *C) {
    hipblasStatus_t status;

#ifdef DOUBLEANN
    status = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, k, &alpha, A, m, B, n, &beta, C, m);
#else
    status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, k, &alpha, A, m, B, n, &beta, C, m);
#endif
    if (status != HIPBLAS_STATUS_SUCCESS) {
        HError(8890, (char *)"HNBlasNTgemmCUDA: CUBLAS library gemm function failed");
    }
}

/*  */
void HNBlasTNgemmCUDA(int m, int n, int k, NFloat alpha, NFloat *A, NFloat *B, NFloat beta, NFloat *C) {
    hipblasStatus_t status;

#ifdef DOUBLEANN
    status = hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, k, &alpha, A, k, B, k, &beta, C, m);
#else
    status = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, k, &alpha, A, k, B, k, &beta, C, m);
#endif
    if (status != HIPBLAS_STATUS_SUCCESS) {
        HError(8890, (char *)"HNBlasTNgemmCUDA: CUBLAS library gemm function failed");
    }
}

/*  */
void CalXENTCriterionCUDA(NFloat *refPtr, NFloat *hypPtr, int segLen, NFloat *crtPtr) {
    HKern_CalXENTCriterionCUDA<<<1, THREADPERBLOCK>>>(refPtr, hypPtr, segLen, crtPtr);
}

/*  */
void CalMMSECriterionCUDA(NFloat *refPtr, NFloat *hypPtr, int segLen, NFloat *crtPtr) {
    HKern_CalMMSECriterionCUDA<<<1, THREADPERBLOCK>>>(refPtr, hypPtr, segLen, crtPtr);
}

/*  */
void AddNSegmentTargetPenCUDA(NFloat *srcSeg, NFloat *penSeg, int row, int col, NFloat *dstSeg) {
    int nBlocks, size;

    size = row * col;
    nBlocks = CEIL(size, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"AddNVectorTargetPenCUDA: Block number exceeds the maximum");

    HKern_AddSegmentTargetPen<<<nBlocks, THREADPERBLOCK>>>(srcSeg, penSeg, row, col, dstSeg);
}

void FindMaxElementCUDA(NFloat *srcPtr, int row, int col, NFloat *dstPtr) {
    int nBlocks, sBytes;

    nBlocks = row;
    sBytes = 2 * sizeof(NFloat) * THREADPERBLOCK;
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"FindMaxElementCUDA: Block number exceeds the maximum");
    HKern_RedMaxElementIndex<<<nBlocks, THREADPERBLOCK, sBytes>>>(srcPtr, row, col, dstPtr);
}

/*  */
/*void SubNSegmentByConstCUDA(NFloat *srcSeg, int segLen, NFloat constVal, NFloat *dstSeg) {
    int nBlocks;

    nBlocks = CEIL(segLen, THREADPERBLOCK); 
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, (char *)"SubNSegmentByConstCUDA: Block number exceeds the maximum");

    HKern_SubNSegmentByConst<<<nBlocks, THREADPERBLOCK>>>(srcSeg, segLen, constVal, dstSeg);
}*/

/* cz277 - semi */
/*  */
void ShiftNSegmentValsCUDA(NFloat *srcSeg, int segLen, NFloat shiftVal, NFloat *dstSeg) {
    int nBlocks;

    nBlocks = CEIL(segLen, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"ShiftNSegmentValsCUDA: Block number exceeds the maximum");

    HKern_ShiftNSegmentVals<<<nBlocks, THREADPERBLOCK>>>(srcSeg, segLen, shiftVal, dstSeg);
}

/* cz277 - 1007 */
void CopyPartialNSegmentCUDA(int minRow, int minCol, NFloat *srcPtr, int srcCol, NFloat *dstPtr, int dstCol) {
    int len, nBlocks;

    len = minRow * minCol;
    nBlocks = CEIL(len, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"CopyPartialNSegmentCUDA: Block number exceeds the maximum");
    HKern_CopyPartialNSegment<<<nBlocks, THREADPERBLOCK>>>(minRow, minCol, srcPtr, srcCol, dstPtr, dstCol);
}

/* --------------------------- HFBLat funcs ------------------------ */

/* cz277 - cuda fblat */
void SetModelBetaPlusCUDA(int T, NMatrix *llhMat, int *qLo, int *qHi, int Q, float probScale, AcousticDev *acList) {
    int nBlocks;

    /* t in [1 ... T]; q in [1 ... Q] */
    nBlocks = CEIL(T * Q, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"SetModelBetaPlusCUDA: Block number exceeds the maximum");
    /* setotprob */
    HKern_Setotprob4q<<<nBlocks, THREADPERBLOCK>>>(T, llhMat->devElems, llhMat->colNum, qLo, qHi, Q, probScale, acList);
    /* set model beta plus */
    nBlocks = CEIL(Q, THREADPERBLOCK);
    HKern_SetModelPlus<<<nBlocks, THREADPERBLOCK>>>(Q, acList);

} 


/* cz277 - cuda fblat */
void ZeroAlphasCUDA(int T, int Q, AcousticDev *acList) {
    int nBlocks;

    nBlocks = CEIL(T * Q, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"ZeroAlphasCUDA: Block number exceeds the maximum");
    HKern_ZeroAlphas<<<nBlocks, THREADPERBLOCK>>>(T, Q, acList);
}


/* cz277 - cuda fblat */
void StepAlphaCUDA(int Q, AcousticDev *acList) {
    int nBlocks;

    nBlocks = CEIL(Q, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"StepAlphaCUDA: Block number exceeds the maximum");
    HKern_StepAlpha<<<nBlocks, THREADPERBLOCK>>>(Q, acList);
}

/* cz277 - gradlim */
void ClipNSegmentValsCUDA(NFloat* srcSeg, int len, NFloat upperLim, NFloat lowerLim, NFloat *dstSeg) {
    int nBlocks;

    nBlocks = CEIL(len, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"LimitNSegmentValsCUDA: Block number exceeds the maximum");
    HKern_ClipNSegmentVals<<<nBlocks, THREADPERBLOCK>>>(srcSeg, len, upperLim, lowerLim, dstSeg);
}

/* cz277 - max norm */
void CalNMatrixL2NormByRowCUDA(NFloat *matPtr, int row, int col, NFloat *normPtr) {
    int nBlocks, sBytes;
  
    nBlocks = row;
    sBytes = sizeof(NFloat) * THREADPERBLOCK;
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"CalExtNMatrixL2NormCUDA: Block number exceeds the maximum");
    HKern_RedCalNMatrixL2NormByRow<<<nBlocks, THREADPERBLOCK, sBytes>>>(matPtr, row, col, normPtr);
}

void DivideNMatrixByRowCUDA(NFloat *srcPtr, int row, int col, NFloat *normPtr, NFloat *dstPtr) {
    int nBlocks, size;

    size = row * col;
    nBlocks = CEIL(size, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(8890, (char *)"DivideNMatrixByRowCUDA: Block number exceeds the maximum");

    HKern_DivideNMatrixByRow<<<nBlocks, THREADPERBLOCK>>>(srcPtr, row, col, normPtr, dstPtr);
}


#ifdef __cplusplus
}
#endif


/* --------------------------- End of HCUDA.cu ---------------------------- */

